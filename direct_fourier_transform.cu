#include "hip/hip_runtime.h"

// Copyright 2019 Adam Campbell, Seth Hall, Andrew Ensor
// Copyright 2019 High Performance Computing Research Laboratory, Auckland University of Technology (AUT)

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// 1. Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.

// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.

// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from this
// software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include <>
#include <numeric>

#include "direct_fourier_transform.h"

// THIS VALUE MUST EQUAL NUMBER OF SOURCES IN FILE
#define NUMBER_OF_SOURCES 5

//IMPORTANT: Modify configuration for target GPU and DFT
void init_config(Config *config)
{
	/// Number of sources to process
	config->num_sources = 1;

	// Toggle whether right ascension should be enabled (observation dependant)
	config->enable_right_ascension = false;

	// Number of visibilities per source
	config->num_visibilities = 1;

	config->num_predicted_vis = 1;

	// Disregard visibility w coordinate during transformation
	config->force_zero_w_term = false;
 
	// Use fixed sources (not from file)
	config->synthetic_sources = false;

	// Use fixed visibilities (not from file)
	config->synthetic_visibilities = false;

	// if using synthetic visibility creation, set this flag to
	// Gaussian distribute random visibility positions
	config->gaussian_distribution_sources = false;

	// Origin of Sources
	config->source_file = "../block_data/sources.csv";

	// Source of Visibilities
	config->vis_src_file    = "../block_data/synthetic_visibilities.csv";

	// Destination for processed visibilities
	config->vis_dest_file 	= "../block_data/visibility_block_output.csv";

	// Dimension of Fourier domain grid
	config->grid_size = 8196.0;

	// Fourier domain grid cell size in radians
	config->cell_size = 6.39708380288950e-6;

	// Frequency of visibility uvw terms
	config->frequency_hz = 100e6;

	// Number of frequencies to sample each visibility across
	config->num_frequencies = 128;

	config->frac_fine_frequency = 0.001;

	// Scalar for visibility coordinates
	config->uv_scale = config->grid_size * config->cell_size;

	// Range for synthetic visibility u coordinates
	config->min_u = -(config->grid_size / 2.0);
	config->max_u = config->grid_size / 2.0;

	// Range for synthetic visibility v coordinates  
	config->min_v = -(config->grid_size / 2.0);
	config->max_v = config->grid_size / 2.0;

	// Range for synthetic visibility w coordinates
	config->min_w = config->min_v / 10;
	config->max_w = config->max_v / 10;

	// Number of CUDA threads per block - this is GPU specific
	config->gpu_max_threads_per_block = 1024;

	// Enables/disables the printing of information during DFT
	config->enable_messages = true;
}

void extract_visibilities(Config *config, Source *sources, Visibility *vis_input_uvw,
	Visibility *vis_predicted, Complex *vis_intensity)
{
	//Allocating GPU memory
	PRECISION3 *d_sources;
	PRECISION3 *d_input_vis_uvw;
	PRECISION3 *d_predicted_vis;
	PRECISION2 *d_intensities;

	int num_visibilities = config->num_visibilities;  
	int num_predicted_vis = config->num_predicted_vis;

	if(config->enable_messages)
		printf(">>> UPDATE: Allocating GPU memory...\n\n");

	//copy the sources to the GPU.
	CUDA_CHECK_RETURN(hipMalloc(&d_sources, sizeof(PRECISION3) * config->num_sources));
	CUDA_CHECK_RETURN(hipMemcpy(d_sources, sources, 
		config->num_sources * sizeof(PRECISION3), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	//copy the visibilities to the GPU
	CUDA_CHECK_RETURN(hipMalloc(&d_input_vis_uvw,  sizeof(PRECISION3) * num_visibilities));
	CUDA_CHECK_RETURN(hipMemcpy(d_input_vis_uvw, vis_input_uvw, 
		num_visibilities * sizeof(PRECISION3), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	CUDA_CHECK_RETURN(hipMalloc(&d_predicted_vis,  sizeof(PRECISION3) * num_predicted_vis));
	hipDeviceSynchronize();

	// Allocate memory on GPU for storing extracted visibility intensities
	CUDA_CHECK_RETURN(hipMalloc(&d_intensities, sizeof(PRECISION2) * num_predicted_vis));
	hipDeviceSynchronize();

	// Define number of blocks and threads per block on GPU
	int max_threads_per_block = min(config->gpu_max_threads_per_block, num_visibilities);
	int num_blocks = (int) ceil((double) num_visibilities / max_threads_per_block);
	dim3 kernel_blocks(num_blocks, 1, 1);
	dim3 kernel_threads(max_threads_per_block, 1, 1);

	if(config->enable_messages)
		printf(">>> UPDATE: Calling DFT GPU Kernel to predict %d visibilities...\n\n", num_predicted_vis);

	//record events for timing kernel execution
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	direct_fourier_transform<<<kernel_blocks, kernel_threads>>>(d_input_vis_uvw, d_predicted_vis,
		d_intensities, config->frac_fine_frequency, num_visibilities, num_predicted_vis, d_sources,
		config->num_sources, config->num_frequencies);
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	if(config->enable_messages)
		printf(">>> UPDATE: DFT GPU Kernel Completed, Time taken %f mS...\n\n",milliseconds);

	CUDA_CHECK_RETURN(hipMemcpy(vis_predicted, d_predicted_vis, 
		num_predicted_vis * sizeof(PRECISION3), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	CUDA_CHECK_RETURN(hipMemcpy(vis_intensity, d_intensities, 
		num_predicted_vis * sizeof(PRECISION2), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	if(config->enable_messages)
		printf(">>> UPDATE: Copied Visibility Data back to Host - Completed...\n\n");

	// Clean up
	CUDA_CHECK_RETURN(hipFree(d_sources));
	CUDA_CHECK_RETURN(hipFree(d_input_vis_uvw));
	CUDA_CHECK_RETURN(hipFree(d_predicted_vis));
	CUDA_CHECK_RETURN(hipFree(d_intensities));
	CUDA_CHECK_RETURN(hipDeviceReset());
}

__device__ double2 complex_mult(const double2 z1, const double2 z2)
{
	return make_double2(z1.x * z2.x - z1.y * z2.y, z1.y * z2.x + z1.x * z2.y);
}

__global__ void direct_fourier_transform(const PRECISION3 *d_vis_uvw, PRECISION3 *d_predicted_vis, 
	PRECISION2 *d_intensities, const double frac_fine_frequency, const int num_vis, const int num_predicted_vis, 
	const PRECISION3 *sources, const int num_sources, const int num_frequencies)
{
	const int vis_indx = blockIdx.x * blockDim.x + threadIdx.x;

	if(vis_indx >= num_vis)
		return;

	PRECISION2 source_sum = MAKE_PRECISION2(0.0, 0.0);
	PRECISION term = 0.0;
	PRECISION w_correction = 0.0;
	PRECISION image_correction = 0.0;
	PRECISION theta = 0.0;
	PRECISION src_correction = 0.0;

	const PRECISION3 vis = d_vis_uvw[vis_indx];
	PRECISION3 src;
	PRECISION2 theta_complex = MAKE_PRECISION2(0.0, 0.0);

	PRECISION3 uvw_delta = MAKE_PRECISION3(
		d_vis_uvw[vis_indx].x * frac_fine_frequency,
		d_vis_uvw[vis_indx].y * frac_fine_frequency,
		d_vis_uvw[vis_indx].z * frac_fine_frequency
	);

	PRECISION2 scale_freq[NUMBER_OF_SOURCES];
	PRECISION2 current_freq[NUMBER_OF_SOURCES];

	const double two_PI = HIP_PI + HIP_PI;
	// For all sources
	for(int src_index = 0; src_index < num_sources; ++src_index)
	{	
		src = sources[src_index];
		
		// square root formula (most accurate method)
		// term = sqrt(1.0 - (src.x * src.x) - (src.y * src.y));
		// image_correction = term;
		// w_correction = term - 1.0; 

		// approximation formula (unit test fails as less accurate)
		term = 0.5 * ((src.x * src.x) + (src.y * src.y));
		w_correction = -term;
		image_correction = 1.0 - term;

		src_correction = src.z / image_correction;

		theta = (vis.x * src.x + vis.y * src.y + vis.z * w_correction) * two_PI;
		sincos(theta, &(theta_complex.y), &(theta_complex.x));
		current_freq[src_index].x = theta_complex.x * src_correction;
		current_freq[src_index].y = -theta_complex.y * src_correction;

		theta = (uvw_delta.x * src.x + uvw_delta.y * src.y + uvw_delta.z * w_correction) * two_PI;
		sincos(theta, &(theta_complex.y), &(theta_complex.x));
		scale_freq[src_index] = MAKE_PRECISION2(theta_complex.x, -theta_complex.y);
	}

	for(int freq_index = 0; freq_index < num_frequencies; ++freq_index)
	{
		PRECISION2 current_vis = MAKE_PRECISION2(0.0, 0.0);

		for(int src_index = 0; src_index < num_sources; ++src_index)
		{
			current_vis.x += current_freq[src_index].x;
		 	current_vis.y += current_freq[src_index].y;

		 	current_freq[src_index] = complex_mult(current_freq[src_index], scale_freq[src_index]);
		}

		int strided_vis_index = (freq_index * num_vis) + vis_indx;

		d_intensities[strided_vis_index] = current_vis;
		d_predicted_vis[strided_vis_index] = MAKE_PRECISION3(
			d_vis_uvw[vis_indx].x + freq_index * uvw_delta.x,
			d_vis_uvw[vis_indx].y + freq_index * uvw_delta.y,
			d_vis_uvw[vis_indx].z + freq_index * uvw_delta.z
		);
	}
}

void load_visibilities(Config *config, Visibility **vis_input_uvw, Visibility **predicted_vis,
	Complex **vis_intensity)
{
	if(config->enable_messages)
		printf(">>> UPDATE: Using Visibilities from file...\n\n");

	FILE *file = fopen(config->vis_src_file, "r");
	if(file == NULL)
	{
		printf(">>> ERROR: Unable to locate visibilities file...\n\n");
		return;
	}

	// Reading in the counter for number of visibilities
	fscanf(file, "%d\n", &(config->num_visibilities));
	config->num_predicted_vis = config->num_visibilities * config->num_frequencies;
	*vis_input_uvw = (Visibility*) calloc(config->num_visibilities, sizeof(Visibility));
	*predicted_vis = (Visibility*) calloc(config->num_predicted_vis, sizeof(Visibility));
	*vis_intensity =  (Complex*) calloc(config->num_predicted_vis, sizeof(Complex));

	// File found, but was memory allocated?
	if(*vis_input_uvw == NULL || *predicted_vis == NULL || *vis_intensity == NULL)
	{
		printf(">>> ERROR: Unable to allocate memory for visibilities...\n\n");
		if(file) fclose(file);
		if(*vis_input_uvw) free(*vis_input_uvw);
		if(*predicted_vis) free(*predicted_vis);
		if(*vis_intensity) free(*vis_intensity);
		return;
	}

	double u = 0.0;
	double v = 0.0;
	double w = 0.0;
	double real = 0.0;
	double imag = 0.0;
	double weight = 0.0;

	// Used to scale visibility coordinates from wavelengths
	// to meters
	double wavelength_to_meters = config->frequency_hz / C;
	double right_asc_factor = (config->enable_right_ascension) ? -1.0 : 1.0;

	// Read in n number of visibilities
	for(int vis_indx = 0; vis_indx < config->num_visibilities; ++vis_indx)
	{
		// Read in provided visibility attributes
		// u, v, w, vis real, vis imaginary, weight
		fscanf(file, "%lf %lf %lf %lf %lf %lf\n", &u, &v, &w, 
			&real, &imag, &weight);

		u *=  right_asc_factor;
		w *=  right_asc_factor;

		(*vis_input_uvw)[vis_indx] = (Visibility) {
			.u = u * wavelength_to_meters,
			.v = v * wavelength_to_meters,
			.w = (config->force_zero_w_term) ? 0.0 : w * wavelength_to_meters
		};
	}

	// Clean up
	fclose(file);

	if(config->enable_messages)
		printf(">>> UPDATE: Successfully loaded %d visibilities from file...\n\n",config->num_visibilities);
}

void load_sources(Config *config, Source **sources)
{
	if(config->enable_messages)
		printf(">>> UPDATE: Using Sources from file...\n\n");

	FILE *file = fopen(config->source_file, "r");
	// Unable to open file
	if(file == NULL)
	{	
		printf(">>> ERROR: Unable to load sources from file...\n\n");
		return;
	}

	fscanf(file, "%d\n", &(config->num_sources));

	if(config->num_sources != NUMBER_OF_SOURCES)
	{
		printf(">>> ERROR: Number of sources from file does not match #define value!!!\n");
		exit(EXIT_FAILURE);
	}

	*sources = (Source*) calloc(config->num_sources, sizeof(Source));
	if(*sources == NULL)
 	{
 		fclose(file);
 		return;
	}

	PRECISION temp_l = 0.0;
	PRECISION temp_m = 0.0;
	PRECISION temp_intensity = 0.0;

	for(int src_indx = 0; src_indx < config->num_sources; ++src_indx)
	{
		fscanf(file, "%lf %lf %lf\n", &temp_l, &temp_m, &temp_intensity);

		(*sources)[src_indx] = (Source) {
			.l = temp_l * config->cell_size,
			.m = temp_m * config->cell_size,
			.intensity = temp_intensity
		};
	}

	// Clean up
	fclose(file);
	if(config->enable_messages)
		printf(">>> UPDATE: Successfully loaded %d sources from file..\n\n",config->num_sources);
}


void save_visibilities(Config *config, Visibility *predicted, Complex *intensities)
{
	// Save visibilities to file
	FILE *file = fopen(config->vis_dest_file, "w");
	// Unable to open file
	if(file == NULL)
	{
		printf(">>> ERROR: Unable to save visibilities to file...\n\n");
		return;
	}

	if(config->enable_messages)
		printf(">>> UPDATE: Writing visibilities to file...\n\n");

	// Record number of visibilities
	fprintf(file, "%d\n", config->num_predicted_vis);
	
	// Used to scale visibility coordinates from meters to
	// wavelengths (useful for gridding, inverse DFT etc.)
	double meters_to_wavelengths = config->frequency_hz / C;

	// Record individual visibilities
	for(int vis_indx = 0; vis_indx < config->num_predicted_vis; ++vis_indx)
	{

		predicted[vis_indx].u /= meters_to_wavelengths;
		predicted[vis_indx].v /= meters_to_wavelengths;
		predicted[vis_indx].w /= meters_to_wavelengths;

		if(config->enable_right_ascension)
		{
			predicted[vis_indx].u *= -1.0;
			predicted[vis_indx].w *= -1.0;
		}

		// u, v, w, real, imag, weight (intensity)
		fprintf(file, "%f %f %f %f %f %f\n", 
			predicted[vis_indx].u,
			predicted[vis_indx].v,
			predicted[vis_indx].w,
			intensities[vis_indx].real,
			intensities[vis_indx].imaginary,
			1.0); // static intensity (for now)
	}

	// Clean up
	fclose(file);
	if(config->enable_messages)
		printf(">>> UPDATE: Completed writing of visibilities to file...\n\n");
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void check_cuda_error_aux(const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;

	printf(">>> CUDA ERROR: %s returned %s at %s : %u ",statement, file, hipGetErrorString(err), line);
	exit(EXIT_FAILURE);
}

PRECISION random_in_range(PRECISION min, PRECISION max)
{
	PRECISION range = (max - min);
	PRECISION div = RAND_MAX / range;
	return min + (rand() / div);
}

PRECISION generate_sample_normal()
{
	PRECISION u = ((PRECISION) rand() / RAND_MAX) * 2.0 - 1.0;
	PRECISION v = ((PRECISION) rand() / RAND_MAX) * 2.0 - 1.0;
	PRECISION r = u * u + v * v;
	if(r <= 0.0 || r > 1.0)
		return generate_sample_normal();
	return r * sqrt(-2.0 * log(r) / r);
}

//**************************************//
//      UNIT TESTING FUNCTIONALITY      //
//**************************************//

// void unit_test_init_config(Config *config)
// {
// 	config->num_sources 					= 1;
// 	config->num_visibilities 				= 1;
// 	config->source_file 					= "../unit_test_data/20_synth_sources.csv";
// 	config->vis_src_file    				= "../unit_test_data/1k_vis_input.csv";
// 	config->vis_dest_file 					= "../unit_test_data/1k_vis_output.csv";
// 	config->synthetic_sources 				= false;
// 	config->synthetic_visibilities 			= false;
// 	config->gaussian_distribution_sources 	= false;
// 	config->force_zero_w_term 				= false;
// 	config->enable_right_ascension			= false;
// 	config->grid_size 						= 18000;
// 	config->cell_size 						= 6.39708380288950e-6;
// 	config->frequency_hz 					= 100e6;
// 	config->uv_scale 						= config->grid_size * config->cell_size;
// 	config->min_u 							= -(config->grid_size / 2.0);
// 	config->max_u 							= config->grid_size / 2.0;
// 	config->min_v 							= -(config->grid_size / 2.0);
// 	config->max_v 							= config->grid_size / 2.0;
// 	config->min_w 							= config->min_v / 10;
// 	config->max_w 							= config->max_v / 10;
// 	config->gpu_max_threads_per_block		= 1;
// 	config->enable_messages 				= false;
// }

// double unit_test_generate_approximate_visibilities(void)
// {
// 	// used to invalidate the unit test
// 	double error = DBL_MAX;

// 	Config config;
// 	unit_test_init_config(&config);

// 	// Read in test sources
// 	Source *sources = NULL;
// 	load_sources(&config, &sources);
// 	if(sources == NULL)
// 		return error;

// 	// Read in test visibilities and process
// 	FILE *file = fopen(config.vis_src_file, "r");
// 	if(file == NULL)
// 	{
// 		if(sources) free(sources);
// 		return error;
// 	}

// 	fscanf(file, "%d\n", &(config.num_visibilities));

// 	double u = 0.0;
// 	double v = 0.0;
// 	double w = 0.0;
// 	double intensity = 0.0;
// 	double difference = 0.0;
// 	double wavelength_to_meters = config.frequency_hz / C;
// 	Complex brightness = (Complex) {.real = 0.0, .imaginary = 0.0};
// 	Complex test_vis_intensity;
// 	Visibility approx_visibility[1]; // testing one at a time
// 	Complex approx_vis_intensity[1]; // testing one at a time

// 	for(int vis_indx = 0; vis_indx < config.num_visibilities; ++vis_indx)
// 	{
// 		fscanf(file, "%lf %lf %lf %lf %lf %lf\n", &u, &v, &w, 
// 			&(brightness.real), &(brightness.imaginary), &intensity);

// 		test_vis_intensity.real      = brightness.real;
// 		test_vis_intensity.imaginary = brightness.imaginary;

// 		approx_visibility[0] = (Visibility) {
// 			.u = u * wavelength_to_meters,
// 			.v = v * wavelength_to_meters,
// 			.w = w * wavelength_to_meters
// 		};

// 		approx_vis_intensity[0] = (Complex) {
// 			.real      = 0.0,
// 			.imaginary = 0.0
// 		};

// 		// Measure one visibility brightness from n sources
// 		extract_visibilities(&config, sources, approx_visibility, approx_vis_intensity, 1);

// 		double current_difference = sqrt(pow(approx_vis_intensity[0].real
// 			-test_vis_intensity.real, 2.0)
// 			+ pow(approx_vis_intensity[0].imaginary
// 			-test_vis_intensity.imaginary, 2.0));

// 		if(current_difference > difference)
// 			difference = current_difference;
// 	}

// 	// Clean up
// 	fclose(file);
// 	if(sources) free(sources);

// 	printf(">>> INFO: Measured maximum difference of evaluated visibilities is %f\n", difference);

// 	return difference;
// }
